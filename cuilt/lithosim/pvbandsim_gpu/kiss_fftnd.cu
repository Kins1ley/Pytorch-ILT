#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>          //CUFFT文件头

// Helper functions for CUDA
#include ""

#include <stdio.h>
#include "lithosim/pvbandsim_gpu/kiss_fft.h"
//#include "kiss_fft.h"

#define NX 2048
#define NY 2048



void kiss_fftnd_cuda(const kiss_fft_cpx *fin, kiss_fft_cpx *fout,int isinverse){
    hipfftComplex *idata, *odata;   //显存数据指针
    //在显存中分配空间
    hipMalloc((void**)&idata, sizeof(hipfftComplex)*NX*NY);
    hipMalloc((void**)&odata, sizeof(hipfftComplex)*NX*NY);

    if(hipGetLastError() != hipSuccess){
        fprintf(stderr, "Cuda error: Failed to allocate\n");
        return;
    }
    hipMemcpy(idata, fin, sizeof(hipfftComplex)*NX*NY, hipMemcpyHostToDevice);
    /*
    Do not use for loop to assign the value
    */
    //创建CUFFT句柄
    hipfftHandle plan;
    hipfftPlan2d(&plan, NX, NY, HIPFFT_C2C);

    //执行CUFFT
    if(isinverse){
        // printf("now excute the inverse fft \n");
        hipfftExecC2C(plan, idata, odata, HIPFFT_BACKWARD);  //快速傅里叶反向变换
    }else{
        // printf("now excute the forward fft \n");
        hipfftExecC2C(plan, idata, odata, HIPFFT_FORWARD);  //快速傅里叶正变换
    }

    /*
    Still, Do not use for loop to assign the value.
    Use hipMemcpy
    */
    hipMemcpy(fout, odata, sizeof(hipfftComplex)*NX*NY, hipMemcpyDeviceToHost);
    //销毁句柄，并释放空间
    hipfftDestroy(plan);
    hipFree(idata);
    hipFree(odata);
}


