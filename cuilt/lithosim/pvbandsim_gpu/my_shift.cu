#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>          //CUFFT文件头

// Helper functions for CUDA
#include ""

#include <stdio.h>
#include "lithosim/pvbandsim_gpu/kiss_fft.h"
#include "lithosim/pvbandsim_gpu/cufftShift.h"
//#include "kiss_fft.h"
//#include "cufftShift.h"

#define NX 2048
#define NY 2048

void my_shift_cuda(const kiss_fft_cpx *fin, kiss_fft_cpx *fout){
    hipfftComplex *idata, *odata;   //显存数据指针
    //在显存中分配空间
    hipMalloc((void**)&idata, sizeof(hipfftComplex)*NX*NY);
    hipMalloc((void**)&odata, sizeof(hipfftComplex)*NX*NY);
    if(hipGetLastError() != hipSuccess){
        fprintf(stderr, "Cuda error: Failed to allocate\n");
        return;
    }
    hipMemcpy(idata, fin, sizeof(hipfftComplex)*NX*NY, hipMemcpyHostToDevice);
    cufftShift_2D_impl(idata, odata, NX, NY);
    hipMemcpy(fout, odata, sizeof(hipfftComplex)*NX*NY, hipMemcpyDeviceToHost);
    //销毁句柄，并释放空间
    hipFree(idata);
    hipFree(odata);
}


